#include "hip/hip_runtime.h"
#include "reference_calc.cpp"
#include "utils.h"
#include <stdio.h>

__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{

 int i = blockIdx.x * blockDim.x + threadIdx.x;// determina iteração das linhas
 int j =blockIdx.y * blockDim.y + threadIdx.y; //determina iteração das colunas
    if(i<numRows && j< numCols){
            uchar4 rgba = rgbaImage[i*numCols + j];
            float channelSum = .299f * rgba.x + .587f * rgba.y + .114f * rgba.z;
            greyImage[i*numCols +j]= channelSum;
        
    }
    
    
}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{
  int blockWidth = 32;//Determino qual tamanho de bloco usar
  const dim3 blockSize(blockWidth, blockWidth, 1);  //tamanho do bloco
    int blocksX = numRows/ blockWidth + 1;//qtd de threads
    int blocksY = numCols/ blockWidth + 1;
  const dim3 gridSize( blocksX, blocksY, 1);  //Tamanho do grid
  rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}