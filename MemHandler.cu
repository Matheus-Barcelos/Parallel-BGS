#include "hip/hip_runtime.h"
#include "MemHandler.h"

__global__
void copyImgToGPU(const unsigned char* frameEntrada, const unsigned char* img, int cols, int rows);

__global__
void putInBuffer(const unsigned char* BUFF, int cols, int rows, int sizeBUFF, int pos);

void alloc(unsigned char* &frameEntrada,unsigned char* &frameTratado, unsigned char* &BUFF, int cols, int rows, int sizeBUFF);
void dealloc(unsigned char* &frameEntrada, unsigned char* &frameTratado, unsigned char* &BUFF, int cols, int rows, int sizeBUFF);

