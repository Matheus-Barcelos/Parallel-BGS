#include "hip/hip_runtime.h"
#include "MemHandler.h"

__global__
void copyImgToGPU(const unsigned char* frameEntrada, const unsigned char* img, int cols, int rows);

__global__
void putInBuffer(const unsigned char* BUFF, const unsigned char* frameAtual, int cols, int rows, int sizeBUFF, int pos);

void alloc(unsigned char* &frameEntrada, unsigned char* &frameIntermediario, unsigned char* &frameTratado, unsigned char* &fore, unsigned char* &BUFF, int cols, int rows, int sizeBUFF);
void dealloc(unsigned char* &frameEntrada, unsigned char* &frameIntermediario, unsigned char* &frameTratado, unsigned char* &fore, unsigned char* &BUFF, int cols, int rows, int sizeBUFF);

