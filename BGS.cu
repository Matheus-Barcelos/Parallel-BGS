#include "hip/hip_runtime.h"
#include "BGS.h"
__global__
void BGS(uchar* buffer, int buffer_size, uchar* frame, int frame_size, uchar* frameOut){
	const int x =  blockIdx.x * blockDim.x + threadIdx.x;
    const int y =  blockIdx.y * blockDim.y + threadIdx.y;
    int avg = 0;
    int sum = 0;
    uchar* aux;
    if(x < frame_size && y < frame_size){
    	for(int i = 0; i < buffer_size; i++){
    		aux = buffer[i];
            sum = aux[x][y];
     	}
     	avg = sum / buffer_size;
     	frameOut[x][y] = frame[x][y] - avg;
        //calcula a diferença e escreve no frame do vetor resposta
    }
}
